/*
 * Connection.cpp
 *
 *  Created on: May 30, 2019
 *      Author: sakex
 */

#include "Connection.cuh"

namespace NeuralNetwork {

    Connection::Connection(double const _input_weight, double const _memory_weight, double const riw, double const rmw,
                           double const uiw, double const umw, Neuron *output) :
            input_weight{new double(_input_weight)},
            memory_weight(new double(_memory_weight)),
            reset_input_weight(new double (riw)),
            reset_memory_weight(new double (rmw)),
            update_input_weight(new double (uiw)),
            update_memory_weight(new double (umw)),
            memory(new double(0)),
            prev_input(new double(0)),
            output{output} {
        hipMallocManaged(&memory, 1);
        hipMallocManaged(&prev_input, 1);
        hipMallocManaged(&input_weight, 1);
        hipMallocManaged(&memory_weight, 1);
        hipMallocManaged(&reset_input_weight, 1);
        hipMallocManaged(&reset_memory_weight, 1);
        hipMallocManaged(&update_memory_weight, 1);
        hipDeviceSynchronize();
    }

    Connection::~Connection() {
        hipFree(memory);
        hipFree(prev_input);
        hipFree(input_weight);
        hipFree(memory_weight);
        hipFree(reset_input_weight);
        hipFree(reset_memory_weight);
        hipFree(update_input_weight);
        hipFree(update_memory_weight);
        hipDeviceSynchronize();
    }

    __device__ void Connection::activate(double const value) {
        double const prev_reset = output->get_prev_reset();
        *memory = (*prev_input) * (*input_weight) + (*memory_weight) * (prev_reset) * (*memory);
        *prev_input = value;
        output->increment_memory((*memory) * (*memory_weight));
        output->increment_input(value * *input_weight);
        output->increment_reset(value * *reset_input_weight + *memory * *reset_memory_weight);
        output->increment_update(value * *update_memory_weight + *memory * *update_memory_weight);
    }


} /* namespace NeuralNetwork */
