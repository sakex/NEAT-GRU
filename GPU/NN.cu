#include "hip/hip_runtime.h"
//
// Created by alexandre on 16.06.20.
//

#include "Neuron.cuh"
#include "NN.cuh"

#include "Connection.cuh"

namespace NeuralNetwork {
    __device__ void Connection::init(double const _input_weight, double const _memory_weight, double const riw,
                                     double const rmw,
                                     double const uiw, double const umw, Neuron *_output) {
        memory = 0.f;
        prev_input = 0.f;
        input_weight = _input_weight;
        memory_weight = _memory_weight;
        reset_input_weight = riw;
        reset_memory_weight = rmw;
        update_input_weight = uiw;
        update_memory_weight = umw;
        output = _output;
    }

    __device__ void Connection::activate(double const value) {
        double const prev_reset = output->get_prev_reset();
        memory = prev_input * input_weight + memory_weight * prev_reset * memory;
        prev_input = value;

        double const update_mem = memory * memory_weight;
        output->increment_state(update_mem,
                                value * input_weight,
                                value * reset_input_weight + memory * reset_memory_weight,
                                value * update_input_weight + memory * update_memory_weight);

    }

    __device__ inline void Connection::reset_state() {
        memory = 0.f;
        prev_input = 0.f;
    }
}

namespace NeuralNetwork {
    __device__ inline double fast_sigmoid(double const value) {
        return value / (1.f + std::abs(value));
    }


    __device__ inline double fast_tanh(double const x) {
        if (std::abs(x) >= 4.97) {
            double const values[2] = {-1., 1.};
            return values[x > 0.];
        }
        double const x2 = x * x;
        double const a = x * (135135.0f + x2 * (17325.0f + x2 * (378.0f + x2)));
        double const b = 135135.0f + x2 * (62370.0f + x2 * (3150.0f + x2 * 28.0f));
        return a / b;
    }

    __device__ void
    Neuron::add_connection(Neuron *neuron, double const input_weight, double const memory_weight, double const riw,
                           double const rmw,
                           double const uiw, double const umw) {
        Connection *co = &connections[last_connection_added++];
        co->init(input_weight, memory_weight, riw, rmw, uiw, umw, neuron);
    }

    __device__ void Neuron::init() {
        connections = nullptr;
        input = 0.;
        memory = 0.;
        update = 0.;
        reset = 0.;
        prev_reset = 0.;
        last_connection_added = 0;
    }

    __device__ inline void
    Neuron::increment_state(double const mem, double const inp, double const res, double const upd) {
        memory += mem;
        input += inp;
        reset += res;
        update += upd;
    }

    __device__ double Neuron::get_prev_reset() const {
        return prev_reset;
    }

    __device__ void Neuron::reset_value() {
        input = 0.;
        update = 0.;
        memory = 0.;
    }

    __device__ void Neuron::set_input_value(double new_value) {
        input = new_value;
    }

    __device__ double Neuron::get_value() {
        const double update_gate = fast_sigmoid(update);
        const double reset_gate = fast_sigmoid(reset);
        const double current_memory = fast_tanh(input + memory * reset_gate);
        const double value = update_gate * memory + (1.f - update_gate) * current_memory;
        prev_reset = reset_gate;
        reset_value();
        return fast_tanh(value);
    }

    __device__ void Neuron::free_connections() {
        delete[]connections;
        connections = nullptr;
    }

    __device__ inline void Neuron::set_connections_count(size_t count) {
        connections = new Connection[count]();
    }

    __device__ inline void Neuron::feed_forward() {
        double const update_gate = fast_sigmoid(update);
        double const reset_gate = fast_sigmoid(reset);

        const double current_memory = fast_tanh(input + memory * reset_gate);
        const double value = update_gate * memory + (1.f - update_gate) * current_memory;
        for (int i = 0; i < last_connection_added; ++i) {
            connections[i].activate(value);
        }
        prev_reset = reset_gate;
        reset_value();
    }

    __device__ inline void Neuron::reset_state() {
        reset_value();
        prev_reset = 0.;
        for (int i = 0; i < last_connection_added; ++i) {
            connections[i].reset_state();
        }
    }
}

namespace NeuralNetwork {

    size_t NN::current_id = 0;

    NN::NN() :
            stream{nullptr},
            neurons_count(0),
            layers{nullptr},
            layer_count(0),
            layer_addresses{nullptr} {
        id = current_id++;
        hipError_t err = hipStreamCreate(&stream);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
    }

    NN::NN(Topology &topology) :
            stream{nullptr},
            neurons_count(0),
            layers(nullptr),
            layer_count(0),
            layer_addresses{nullptr} {
        id = current_id++;
        hipError_t err = hipStreamCreate(&stream);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
        init_topology(topology);
    }

    NN::~NN() {
        delete[] layer_addresses;
        delete_layers();
        hipError_t err = hipStreamDestroy(stream);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
    }

    __global__ void free_connections_kernel(Neuron *neurons) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        neurons[tid].free_connections();
    }

    void NN::delete_layers() {
        free_connections_kernel<<<1, neurons_count, id, stream>>>(layers);
        hipError_t err = hipFree(layers);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
    }

    __global__ void init_kernel(Neuron *neurons) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        neurons[tid].init();
    }

    __global__ void set_connections_kernel(Neuron *layers, CUDAConnectionCount *connection_count) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        CUDAConnectionCount *count = &connection_count[tid];
        Neuron *input_neuron_ptr = &layers[count->pos];
        input_neuron_ptr->set_connections_count(count->count);
    }

    __global__ void connect_neurons_kernel(Neuron *layers, CUDAGene *genes, size_t N) {
        for (size_t it = 0; it < N; ++it) {
            CUDAGene *phen = &genes[it];
            Neuron *input_neuron_ptr = &layers[phen->input_pos];
            Neuron *output_neuron_ptr = &layers[phen->output_pos];

            input_neuron_ptr->add_connection(
                    output_neuron_ptr,
                    phen->input_weight,
                    phen->memory_weight,
                    phen->reset_input_weight,
                    phen->reset_memory_weight,
                    phen->update_input_weight,
                    phen->update_memory_weight);
        }
    }

    void NN::init_topology(Topology &topology) {
        layer_count = topology.get_layers();
        std::vector<int> const &sizes = topology.get_layers_size();
        layer_addresses = new int[layer_count + 1];
        neurons_count = 0;
        int i = 0;
        for (; i < layer_count; ++i) {
            layer_addresses[i] = neurons_count;
            neurons_count += sizes[i];
        }
        layer_addresses[i] = neurons_count;
        hipError_t err = hipMalloc(&layers, sizeof(Neuron) * neurons_count);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
        init_kernel<<<1, neurons_count, id, stream>>>(layers);
        Topology::relationships_map &relationships = topology.get_relationships();
        std::vector<CUDAGene> gene_vec;
        std::vector<CUDAConnectionCount> connection_counts;

        for (auto &it : relationships) {
            connection_counts.push_back({layer_addresses[it.first[0]] + it.first[1],
                                         it.second.genes.size()}
            );

            for (Gene *gene : it.second.genes) {
                if (gene->is_disabled()) {
                    continue;
                }
                Gene::point input = gene->get_input();
                Gene::point output = gene->get_output();
                gene_vec.push_back({
                                           gene->get_input_weight(),
                                           gene->get_memory_weight(),
                                           gene->get_reset_input_weight(),
                                           gene->get_reset_memory_weight(),
                                           gene->get_update_input_weight(),
                                           gene->get_update_memory_weight(),
                                           layer_addresses[input[0]] + input[1],
                                           layer_addresses[output[0]] + output[1]
                                   });
            }
        }
        CUDAConnectionCount *device_counts;
        err = hipMalloc(&device_counts, sizeof(CUDAConnectionCount) * connection_counts.size());

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }

        err = hipMemcpyAsync(device_counts, connection_counts.data(),
                              sizeof(CUDAConnectionCount) * connection_counts.size(),
                              hipMemcpyHostToDevice, stream);

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }

        set_connections_kernel<<<1, connection_counts.size(), id, stream>>>(layers, device_counts);

        err = hipFree(device_counts);

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }

        CUDAGene *device_genes;
        err = hipMalloc(&device_genes, sizeof(CUDAGene) * gene_vec.size());

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }

        err = hipMemcpyAsync(device_genes, gene_vec.data(), sizeof(CUDAGene) * gene_vec.size(),
                              hipMemcpyHostToDevice, stream);

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }

        connect_neurons_kernel<<<1, 1, id, stream>>>(layers, device_genes, gene_vec.size());
        err = hipFree(device_genes);

        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
    }

    __device__ double *NN::compute(const double *inputs_array,
                                   size_t const from,
                                   size_t const to,
                                   size_t const output_size,
                                   double *out,
                                   size_t write_from) {
        set_inputs(inputs_array, from, to);
        for (int it = 0; it < neurons_count - output_size; ++it) {
            layers[it].feed_forward();
        }
        for (size_t it = neurons_count - output_size; it < neurons_count; ++it) {
            out[it - neurons_count + output_size] = layers[it].get_value();
        }
        // softmax(out, output_size);
        return out;
    }

    __device__ void NN::reset_state() {
        for (int it = 0; it < neurons_count; ++it) {
            layers[it].reset_state();
        }
    }

    __device__ void NN::set_inputs(const double *inputs_array, size_t const from, size_t const to) {
        for (int i = from; i < to; ++i) {
            layers[i].set_input_value(inputs_array[i]);
        }
    }

} /* namespace NeuralNetwork */

//
// Created by alexandre on 03.09.20.
//

#include "ComputeInstance.cuh"

ComputeInstance *create_compute_instance(Dim dim) {
    return new ComputeInstance(dim);
}

ComputeInstance::ComputeInstance(Dim _dim) :
        dim(_dim) {
}


void ComputeInstance::set_networks(NeuralNetwork::NN *nets, unsigned long int count) {
    networks = nets;
    networks_count = count;
}

void ComputeInstance::update_dataset(double *host_data) {
    const unsigned int size = dim.x * dim.y * dim.z;
    const unsigned int bytes = size * sizeof(double);

    hipError_t err;
    if (data) {
        err = hipFree(data);
        if (err) {
            std::cout << hipGetErrorString(err) << std::endl;
            throw err;
        }
    }

    err = hipMalloc((double **) &data, bytes);
    if (err) {
        std::cout << hipGetErrorString(err) << std::endl;
        throw err;
    }
    err = hipMemcpy(data, host_data, bytes, hipMemcpyHostToDevice);

    if (err) {
        std::cout << hipGetErrorString(err) << std::endl;
        throw err;
    }
}

__global__ void
compute_kernel(Dim dim,
               NeuralNetwork::NN *networks,
               double *data,
               const unsigned long int networks_count,
               const unsigned long int output_size,
               double *d_output) {
    unsigned int id = blockDim.x * gridDim.x;
    if (id < networks_count) {
        NeuralNetwork::NN *net = &networks[id];
        // Number of datasets
        for (size_t i = 0; i < dim.z; ++i) {
            // Size of each dataset
            for (size_t j = 0; j < dim.y; ++j) {
                size_t const from = j * dim.x + i * dim.y;
                size_t const write_from = j * output_size + i * dim.y;
                net->compute(data, from, from + dim.x, output_size, d_output, write_from);
            }
        }
        net->reset_state();
    }
}

void ComputeInstance::compute(const unsigned int output_size) {
    const unsigned int size = dim.y * dim.z * output_size * networks_count;
    const unsigned int bytes = size * sizeof(double);
    h_output = (double *) malloc(bytes);
    hipError_t err = hipMalloc((double **) &d_output, bytes);
    if (err) {
        std::cout << hipGetErrorString(err) << std::endl;
        throw err;
    }
    compute_kernel<<<1, networks_count>>>(dim, networks, data, networks_count, output_size, d_output);
    err = hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);
    if (err) {
        std::cout << hipGetErrorString(err) << std::endl;
        throw err;
    }
    err = hipFree(d_output);
    if (err) {
        std::cout << hipGetErrorString(err) << std::endl;
        throw err;
    }
}

double *ComputeInstance::get_output() const {
    return h_output;
}
